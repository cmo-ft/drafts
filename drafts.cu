#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>
#include <hip/hip_cooperative_groups.h>
#include "MyCudaToolkit.h"
using namespace std;
namespace cg = cooperative_groups;

__device__ unsigned int reduce_sum(long in, cg::thread_block cta)
{
	extern __shared__ long sdata[];


	// Perform first level of reduction:
	// - Write to shared memory
	unsigned int ltid = threadIdx.x;
//printf("1");
	sdata[ltid] = in;

	cg::sync(cta);
//printf("2");
	// Do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (ltid < s)
		{
//printf("%d\n",ltid + s);
			sdata[ltid] += sdata[ltid + s];
		}

		cg::sync(cta);
	}


	return sdata[0];
}

__global__ void SimulateAbsorption(long* count,long size, int flag) {
	cg::thread_block cta = cg::this_thread_block();
	int bid = blockIdx.x;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid == 0) printf("flag %d processing...\n", flag);
	//if (tid > size) return;
	int count_flag = tid % 2;
//printf("1: %d, %d\n", count_flag, threadIdx.x);
	
	if (tid >= size) count_flag = 0;
	count_flag = reduce_sum(count_flag, cta);
	//__syncthreads();

	if (threadIdx.x == 0) {
		//printf("2\n");
		//printf("%d\n", count_flag);
printf("Check sync in GPU\n");
		count[bid] = count_flag;		
	}
if (tid == 0) printf("flag %d complete...\n", flag);

}


int main() {
	long arysize = 1024;
	long* h_count;
	long* d_count;
	
	dim3 grid, block;
	block.x = 1024;
	grid.x = (arysize - 1) / block.x + 1;
	cout << "block.x: " << block.x << "grid.x: " << grid.x << endl;
	h_count = (long*)malloc(grid.x * sizeof(long));
	CHECK(hipMalloc((void**)&d_count, grid.x * sizeof(long)));
printf("Check sync 1. h_count[0]: %d\n", h_count[0]);
	SimulateAbsorption << <grid, block, block.x * sizeof(long) >> > (d_count, arysize, 1);
	//CHECK(hipDeviceSynchronize());
printf("Check sync 2. h_count[0]: %d\n", h_count[0]);
	
CHECK(hipMalloc((void**)&d_count, grid.x * sizeof(long)));
	SimulateAbsorption << <grid, block, block.x * sizeof(long) >> > (d_count, arysize, 2);
printf("Check sync 3. h_count[0]: %d\n", h_count[0]);


	CHECK(hipMemcpy((void*) h_count, (void*) d_count, grid.x * sizeof(long), hipMemcpyDeviceToHost));
	SimulateAbsorption << <grid, block, block.x * sizeof(long) >> > (d_count, arysize, 3);
printf("Check sync 4. h_count[0]: %d\n", h_count[0]);
	
	long total = 0;
	for (int i = 0; i < grid.x; i++) {
		//cout << "element " << i << ": " << h_count[i] << endl;
		total += h_count[i];
	}
	cout << "total: " << total;




	//vector<long*> d_absorb_num_ptrvec, h_absorb_num_ptrvec;
	//d_absorb_num_ptrvec.push_back(nullptr);
	//long* tmplongptr;

	//long** d_absorb_num_ptrary = (long**)malloc(d_absorb_num_ptrvec.size() * sizeof(long*));
	////d_absorb_num_ptrvec[0] = tmplongptr;
	//CHECK(hipMalloc((void**)&d_absorb_num_ptrary[0], grid.x * sizeof(long)));
	////CHECK(hipMalloc((void**)&tmplongptr, grid.x * sizeof(long)));


	return 1;
}